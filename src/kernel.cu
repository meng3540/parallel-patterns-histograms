#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>

__global__ void calculateHisto(char* buffer, int* histo, int size, int numBins)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int sectSize = (size - 1) / (blockDim.x * gridDim.x) + 1;
    int start = i * sectSize;

    extern __shared__ unsigned int histos[];
    for (int binIdx = threadIdx.x; binIdx < numBins; binIdx += blockDim.x) {
        histos[binIdx] = 0u;
    }
    __syncthreads();

    for (int k = 0; k < sectSize; k++) {
        if (start + k < size) {
            int alphaPos = buffer[start + k] - 'a';
            if (alphaPos >= 0 && alphaPos < numBins) {
                atomicAdd(&(histos[alphaPos]), 1);
            }
        }
    }
    __syncthreads();
    for (int binIdx = threadIdx.x; binIdx < numBins; binIdx += blockDim.x) {
        atomicAdd(&(histo[binIdx]),histos[binIdx]);
    }
}

int main()
{
    FILE* file;
    char* buffer;
    long file_length;

    //Allow debugging of input by printing out input text. Very slow on large files so disabled by default.
    int previewInput = 0;

    // Open the file for reading
    file = fopen("enwik8", "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file\n");
        return 1;
    }

    // Get the length of the file
    fseek(file, 0, SEEK_END);
    file_length = ftell(file);
    rewind(file);

    // Allocate memory for the buffer to hold the file content
    buffer = (char*)malloc(file_length * sizeof(char));
    if (buffer == NULL) {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(file);
        return 1;
    }

    // Read the file content into the buffer
    fread(buffer, file_length, 1, file);

    // Close the file
    fclose(file);
    
    //Define and get size of input
    char* input = buffer;

    if (previewInput) {
        printf("Input File:\n");
        for (int i = 0; i < file_length; ++i) {
            printf("%c", buffer[i]);
        }

        printf("\n\n");
    }
    //size_t inputSize = sizeof(input) - 1; // excluding null terminator
    
    //Define number of bins
    int numBins = 26; // number of alphabet letters
    size_t binSize = numBins * sizeof(int);

    //Allocate histogram bins
    int* histo;
    histo = (int*)malloc(binSize);
    for (int i = 0; i < numBins; i++) {
        histo[i] = 0;
    }

    //Define and allocate deviceMemories
    char* deviceInput;
    int* deviceHisto;
    hipMalloc((void**)&deviceInput, file_length);
    hipMalloc((void**)&deviceHisto, binSize);

    //Copy data to device
    hipMemcpy(deviceInput, input, file_length, hipMemcpyHostToDevice);
    hipMemcpy(deviceHisto, histo, binSize, hipMemcpyHostToDevice);

    int blockSize = 32;
    int gridSize = ceil((float)file_length / blockSize); // adjust the gridSize calculation

    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //Kernel Launch
    hipEventRecord(start, 0);
    calculateHisto << < gridSize, blockSize >> > (deviceInput, deviceHisto, file_length, numBins);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(histo, deviceHisto, binSize, hipMemcpyDeviceToHost);

    //Print histogram results
    printf("Histogram:\n");
    for (int i = 0; i < numBins; ++i) {
        printf("%c: %d\n", 'a' + i, histo[i]);
    }

    printf("\n");
    printf("The kernel took %.2f milliseconds to execute.\n", milliseconds);

    hipFree(deviceInput);
    hipFree(deviceHisto);
    free(histo);
    free(buffer);

    return 0;
}
